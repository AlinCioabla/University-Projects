#include "hip/hip_runtime.h"


#include<iostream>
#include<algorithm>
using namespace std;

#define N 25
#define D(row,column) matrix[row*5 + column]

int matrix[N] = {
  0,3,9,8,3,
  5,0,1,4,2,
  6,6,0,4,5,
  2,9,2,0,7,
  7,9,3,2,0,
};

__global__ void RoyFloyd(int* D, int k)
{
  int i = threadIdx.x;
  int j = threadIdx.y;

  D(i, j) = min(D(i, j), D(i, k)) + D(k, j);
}


int main()
{
  int numBlocks = 1;
  int k;
  dim3 threadsPerBlock(N, N);

  int* d_D;
  hipMalloc((void**)&d_D, N * sizeof(int));
  hipMemcpy(&d_D, &matrix, N * sizeof(int), hipMemcpyHostToDevice);

  for (k = 1; k <= N; ++k)
  {
    RoyFloyd<<<numBlocks, threadsPerBlock>>>(d_D, k);
  }
  hipMemcpy(&matrix, &d_D, N * sizeof(int), hipMemcpyDeviceToHost);
}